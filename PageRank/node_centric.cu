#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include<vector>
#include<utility>
#include<algorithm>
#include <hip/hip_runtime.h>

#include <cstdio>
#include <fstream>

using namespace std;

float df = 0.85f;

struct Graph{
    int N; // number of nodes
    int countSink; //number of sink nodes
    int E;
    int *ptrarray;
    int *sinkArray; //sink nodes
    int *adjListArray;  // adjacency list of transposed graph
    int *outdegree; // contains outdegree of all nodes
    int *indegree; //contains indegree of all nodes
    float *pr; // page rank values
};


Graph* buildGraph(vector<pair<int,int>>& edges, int E,int V)
{   
    Graph* G = new Graph();
    G->N = V;
    G->E = E;
    G->ptrarray = new int[V+1];
    G->adjListArray = new int[E]; 
    G-> outdegree = new int[V]();
    G-> indegree = new int[V]();
    G->pr = new float[V];

    
    sort(edges.begin(),edges.end(),[](pair<int,int>& e1,pair<int,int>& e2){
        return (e1.second==e2.second)? (e1.first<e2.first): (e1.second<e2.second);
    });
 
    for(int i=0;i<E;i++){
        G->adjListArray[i] = edges[i].first;
        G->outdegree[edges[i].first]++;
        G->indegree[edges[i].second]++;
    }
 
    int x=0;
    int count = 0;
    for(int i=0;i<=V;i++){
        G->ptrarray[i] = x;
        if (i<V){
            x+= G->indegree[i];
            if (G->outdegree[i] == 0){
                count += 1;
            }
        }
    }

    G-> countSink = count;
    G-> sinkArray = new int[count]();
    x = 0;
    for( int i = 0; i < V; i++){
        if (G->outdegree[i] == 0){
            G-> sinkArray[x] = i;
            x+= 1;
        }
    }
    
    return G;
}


Graph* readgraph(const char* file){

    FILE *in_file = fopen(file, "r");

    int E,V = 0;
    fscanf(in_file, "%d %d", &E,&V);

    vector<pair<int,int>> edges(E);
    for (auto& e:edges) {
        fscanf(in_file, "%d %d", &e.first, &e.second);
    }
    fclose(in_file);
    
    return buildGraph(edges, E, V);
} 

// Stores the page rank values of the given Graph structure in output.txt
void storePageRank(Graph* graph,const char* file)
{
    FILE *out_file = fopen(file, "w");
    for (int i=0; i<graph->N; i++) {
        fprintf(out_file, "%f\n", graph->pr[i]);
    }
    fclose(out_file);
}

// Initialises the page rank values of the Graph structure to 1
void initialisePageRank(Graph *graph)
{

    for (int i=0; i<graph->N; i++) {
        graph->pr[i] = 1.0f;
    }
}


__global__ void  UpdatePagerank(const int* ptrArray,const int* outdegreeArray, const int* sinkArray,const int* adjListArray, 
                                const float* oldpr, float* newpr,int N, float df, int countSink){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    while(idx<N){
        float sum = 0;
        for(int w=ptrArray[idx];w<ptrArray[idx+1];w++){
            int wId = adjListArray[w];
            if (w == ptrArray[idx+1])   //indegree = 0
            break;
            int wOutDegree = outdegreeArray[wId];
            sum += oldpr[wId]/wOutDegree;
        }

        //Add page rank contributed by all sink nodes
        for(int i=0; i<countSink; i++){
            int wId = sinkArray[i];
            sum+= oldpr[wId]/N;
        }
        newpr[idx] = (df*sum) + (1-df)/N;
        idx += gridDim.x*blockDim.x;
    }
}


void PageRank(Graph* G,int iter,float df,int blocksPerGrid,int threadsPerBlock){


    hipError_t err = hipSuccess;

    cout<<"Initialize arrays in device memory\n";
    
    //Allocate the device ptrArray
    int* d_ptrArray = NULL;
    err = hipMalloc((int **)&d_ptrArray, (G->N+1)*sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector ptrArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device ptrArray
    int* d_outdegreeArray = NULL;
    err = hipMalloc((int **)&d_outdegreeArray, (G->N)*sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector outdegreeArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device sinkArray
    int* d_sinkArray = NULL;
    err = hipMalloc((int **)&d_sinkArray, G->countSink * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector ptrArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device adjListArray
    int* d_adjListArray = NULL;
    err = hipMalloc((int **)&d_adjListArray, G->E*sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector adjListArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device oldpr
    float* d_oldpr = NULL;
    err = hipMalloc((float **)&d_oldpr, G->N*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector oldpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device newpr output array
    float* d_newpr = NULL;
    err = hipMalloc((float **)&d_newpr, G->N*sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector newpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //copy input data from host to device
    cout<<"Copy input data from host memory to CUDA device memory\n";
    err = hipMemcpy(d_ptrArray, G->ptrarray, (G->N+1)*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector ptrArray from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_outdegreeArray, G->outdegree, (G->N)*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector outdegree from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_sinkArray, G->sinkArray, G->countSink * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector sinkArray from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_adjListArray, G->adjListArray,  G->E*sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector adjListArray from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_oldpr, G->pr, G->N*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector pr from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // grid and block dimension
    dim3 grid(blocksPerGrid,1,1);
    dim3 block(threadsPerBlock,1,1);
    
    while(iter--){
        // Launch the PageRank Update CUDA Kernel
        UpdatePagerank<<<grid, block>>>(d_ptrArray, d_outdegreeArray, d_sinkArray, d_adjListArray, d_oldpr, 
                                        d_newpr,G->N,df, G->countSink);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch UpdatePageRank kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        //update the oldpr array
        err = hipMemcpy(d_oldpr,d_newpr,G->N*sizeof(float),hipMemcpyDeviceToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy newpr array from device to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    err = hipMemcpy(G->pr,d_oldpr,G->N*sizeof(float),hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy pr array from device to Host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    //free up allocated memory
    err = hipFree(d_ptrArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector ptrArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_adjListArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector adjListArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_oldpr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector oldpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_newpr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector newpr (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_outdegreeArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector outdegreeArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_sinkArray);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector sinkArray (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Reset the device
    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}


int main(){
    char input_file[] = "input.txt";
    char output_file[] = "nodeCentric_output.txt";
    Graph* G = readgraph(input_file);
    initialisePageRank(G);
    int threadsPerBlock = 256;
    int blocksPerGrid = (G->N+threadsPerBlock-1)/threadsPerBlock;
    PageRank(G,1000,df,blocksPerGrid,threadsPerBlock);
    cout<<"PageRank calculation done!!"<<endl;

    storePageRank(G,output_file);
    return 0;
}
