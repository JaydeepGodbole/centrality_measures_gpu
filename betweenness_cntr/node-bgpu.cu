#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

void addEdge(int *adj, int u, int v, int t)
{
    adj[u*t+v] = 1;
    adj[v*t+u] = 1;
}

void printGraph(int *adj, int V)
{
    for (int v = 0; v < V; ++v)
    {
        cout << "\n Adjacency list of vertex "
             << v << "\n head ";
        for (int i = 0; i<V;i++)
        {   
            if(adj[v*V+i]==1)
            {
                cout << "-> " << i;
            }
        }
        printf("\n");
    }
}

__device__ struct ls
{   
    int value;
    ls *next;
};

__device__ ls *pushq(ls *q,int val)
{   
    ls *head = q;
    ls *temp = (ls*)malloc(sizeof(ls));
    
    if(q==NULL)
    {       

        q = (ls*)malloc(sizeof(ls));
        q->value = val;
        q->next = NULL;
        return q;
    }
    else
    {
        while(q->next!=NULL)
        {
            
            q = q->next;
        }
        
        temp->value = val;
        temp->next = NULL;
        q->next = temp;

    }

    return head;
}

__device__ int qfront(ls *q)
{   
    if(q!=NULL)
        return q->value;
    return -1;
}

__device__ int qfrontt(ls *q)
{      
    if(q==NULL)
        return -1;
    while(q->next!=NULL)
    {
        q = q->next;
    }
    return q->value;
}
__device__ ls *qpop(ls *q)
{
    if(q!=NULL)
    {
        ls *temp;// = (ls*)malloc(sizeof(ls));
        temp = q;
        q = q->next;
        free(temp);
        return q;
    }
    return q;
}

__device__ ls *qpopt(ls *q)
{
    if(q!=NULL)
    {
        ls *temp,*head;// = (ls*)malloc(sizeof(ls));
        temp = q;
        head = q;
        ls *fr;
        fr = NULL;
        if(q->next==NULL)
            return NULL;
        while(q->next!=NULL)
        {   
            fr = q;
            q = q->next;
            temp = q;
        }
        fr->next = NULL;
        free(temp);
        return head;
    }
    return q;
}

__device__ void bfs(int src,const int *adj,const int v,
         ls **pred, int *dist, ls **visitstack, int *sigma, float *val)
{
    ls *queue = NULL;
    

    int *visited;
    visited = (int*)malloc(v*sizeof(int));
    if (visited == NULL) 
    {
        return;
    }

    // for (int i = 0; i < v; i++) {
    //     visited[i] = 0;
    //     dist[i] = 1000;
    //     pred[i] = -1;
    // }
    memset(visited, 0, v * sizeof(int));
    memset(dist, 1000, v * sizeof(int));
    //memset(pred, -1, v * sizeof(int));

    visited[src] = 1;
    dist[src] = 0;
    sigma[src] = 1;

    queue = pushq(queue,src);
    
    // standard BFS algorithm
    *val = 0;
    float closeness = 0;
    while (queue!=NULL) {
        int u = qfront(queue);
        queue = qpop(queue);
        *visitstack = pushq(*visitstack,u);
        //printf("Thread %d : queue %d \n",src,qfront(visitstack));
        //break;

        closeness += dist[u];
        for (int j = 0; j < v; j++) 
        {
            if ((visited[j] == 0) && (adj[u*v+j]==1)) 
            {   
                visited[j] = 1;
                dist[j] = dist[u] + 1;
                //pred[j] = u;
                queue = pushq(queue,j);

                // if (j == dest)
                // {
                //     *val = 1;
                //     free(queue);
                //     free(visited);
                //     return;
                // }
            }

            if (dist[j] == dist[u]+1)
            {
                pred[j] = pushq(pred[j],u);
                sigma[j] += sigma[u];
                // free(queue);
                // free(visited);
                //return;
            }
        }
        
    }
    
    if(closeness!=0)
    {
        *val = 1.0/closeness;
    }

    free(visited);
    free(queue);
    return;
}

__global__ void sd(int s,const int *adj, float *score,const int v)
{
    int th = blockDim.x * blockIdx.x + threadIdx.x ;

    if(th<v)
    {       

        //extern __shared__ int sc[];
        //sc[th] = score[th];
        __syncthreads();

        //printf("%d",score[th]);
    	for(int j=0;j<1;j++)
    	{     
            //if(th==249)
                //printf("%d %d\n", th, j);
            // if(j%10==0)
            // {
            //     printf("Hi %d \t %d \t",th,j);
            // }
            if(1)
            {   
                int *dist, *sigma;
                ls **pred;
                float *delta;

                pred = (ls**)malloc(v*sizeof(ls*));
                dist = (int*)malloc(v*sizeof(int));
                sigma = (int*)malloc(v*sizeof(int));
                delta = (float*)malloc(v*sizeof(float));

                if (sigma == NULL) 
                {
                    return;
                }
                if (pred == NULL) 
                {
                    return;
                }
                if (dist == NULL) 
                {
                    return;
                }
                memset(sigma, 0, v * sizeof(int));
                memset(delta, 0.0, v * sizeof(float));

                ls *visitstack=NULL;

                float *val;
                float x = 0;
                val = &x;
                

                
                bfs(th, adj, v, pred, dist, &visitstack, sigma, val);

                //printf("Thread %d : queue %d \n",th,qfront(visitstack));
                while (visitstack!=NULL) 
                {   

                    int w = qfrontt(visitstack);
                    visitstack = qpopt(visitstack);
                    //printf("%d ",w);
                    
                    // For each predecessors of node w, do the math!
                    while(pred[w]!=NULL) 
                    {   
                        int v = qfront(pred[w]);
                        pred[w] = qpop(pred[w]);
                        float c = ((float) sigma[v] / (float) sigma[w]) * (1.0 + delta[w]);
                        //printf("%f",c);
                        delta[v] += c;

                    }
                
                    // Node betweenness aggregation part.
                    if (w != th) 
                    {
                        //score[w] += delta[w];
                        atomicAdd(&score[w],delta[w]);
                    }
                }
                free(visitstack);
                free(pred);
                free(dist);
                free(delta);
            }
            
		    
		}

        
        // = sc[th];
        __syncthreads();
	}

} 

int main()
{
    int V = 250;
    int *adj;
    adj = (int*)malloc(V*V*sizeof(int));

    for(int i=0;i<V;i++)
    {
        for(int j=0;j<V;j++)
        {
            adj[i*V+j] = 0;
        }

    }    
 
    addEdge(adj, 0, 1, V);
    addEdge(adj, 0, 4, V);
    addEdge(adj, 1, 2, V);
    addEdge(adj, 1, 3, V);
    addEdge(adj, 1, 4, V);
    addEdge(adj, 2, 3, V);
    addEdge(adj, 3, 4, V);
    //addEdge(adj, 0, 249, V);
    //printGraph(adj, V);
    /* calculate betweenness centrality score using bfs */

    float *score;
    score = (float*)malloc(V*sizeof(float));
    for (int i = 0; i < V; ++i)
    {
    	score[i] = 0;
    }
 	
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int *adj_A;
    float *result = (float*)malloc(V*sizeof(float));

    err = hipMalloc((void **)&adj_A,V*V*sizeof(int));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *score_A;
    err = hipMalloc((void **)&score_A,V*sizeof(float));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device score A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(adj_A, adj, V*V*sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy adjacent matrix from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(score_A, score, V*sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy score from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    int threadsPerBlock = 256;
    int blocksPerGrid = ((V-1)/256)+1;//(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    sd<<<blocksPerGrid, threadsPerBlock>>>(0,adj_A, score_A, V);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(result, score_A, V*sizeof(int), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(adj_A);
    err = hipFree(score_A);


    for(int j=0;j<10;j++)
    {
     printf("The Score %f \n",(float(result[j])));
    }

    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(result);
    free(score);
    free(adj);


    return 0;
}